#include "hip/hip_runtime.h"


#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#define INITIAL_CAPACITY 1024

/******************** Find the min value **************************/
__global__ void minCompare(int *a, int *set, bool *check, int *capacity) {
    int cap = capacity[0];
    int offset = set[0];
    
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int tabx = idx + cap + offset;
    int taby = idy + cap + offset;

    if (idx == idy) { return; }

    int xval = a[tabx];
    int yval = a[taby];
    
    if(yval <= 1) {}
    else if (xval <= 1) {
        check[idx] = false;
    } else if (xval > yval) {
        check[idx] = false;
    }
}

__global__ void cudaMin(int *a, int *set, bool *check, int* min, int *capacity) {
    int idx = blockIdx.x;

    if (check[idx]) {
        min[0] = a[idx + capacity[0] + set[0]];
    }
}

/************************* Find the max value **********************/
__global__ void maxCompare(int *a, bool *check) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    if (idx == idy) { return; }

    int xval = a[idx];
    int yval = a[idy];

    if (xval < yval) {
        check[idx] = false;
    }
}

__global__ void cudaMax(int *a, bool *check, int* max) {
    int idx = blockIdx.x;

    if (check[idx]) {
        max[0] = a[idx];
    }
}

/*********************** Helper Methods ********************************************/
__global__ void cudaBoolFill(bool *arr, int length) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < length) {
        arr[i] = true;
    }
}

/********************** Min and Max Functions ******************************************/
void findMin(int *arr, const int length, const int offset, int *minimum, int *capacity) {
    //length - 1 = row, offset = location of first element

    bool *check;
    int *min;
    int *set;
    int *row = (int*) malloc(sizeof(int));
    int one = 1;
    const int intSize = sizeof(int);
    const int bsize = length * sizeof(bool);

    hipMalloc((void**)&check, bsize);
    cudaBoolFill<<< dim3(length, 1), 1 >>>(check, length);

    hipMalloc((void**)&min, intSize);
    hipMemcpy(min, (void*)&one, intSize, hipMemcpyHostToDevice);

    hipMalloc((void**)&set, intSize);
    hipMemcpy(set, (int*)&offset, intSize, hipMemcpyHostToDevice);

    hipMemcpy(row, capacity, intSize, hipMemcpyDeviceToHost);
    row[0] = row[0] * (length - 1);
   
    printf("offset = %d    length = %d     row = %d\n", offset, length, row[0]);

    int *row2;
    hipMalloc((void**) &row2, intSize);
    hipMemcpy(row2, row, intSize, hipMemcpyHostToDevice);

    minCompare<<< dim3(length, length), 1 >>>(arr, set, check, row2);
    cudaMin<<< dim3(length, 1), 1 >>>(arr, set, check, min, row2);

    int minhost[1];
    hipMemcpy(minhost, min, intSize, hipMemcpyDeviceToHost);

    hipFree(min);
    hipFree(check);

    hipMemcpy(minimum, (void *)&(minhost[0]), intSize, hipMemcpyHostToDevice);
    //minimum = minhost[0];
}

int findMax(int *arr, const int length) {
    bool *check;
    int *max;

    const int intSize = sizeof(int);
    const int bsize = length * sizeof(bool);

    hipMalloc((void**)&check, bsize);
    cudaBoolFill<<< dim3(length, 1), 1 >>>(check, length);

    hipMalloc((void**)&max, intSize);

    maxCompare<<< dim3(length, length), 1 >>>(arr, check);
    cudaMax<<< dim3(length, 1), 1 >>>(arr, check, max);

    int maxhost[1];
    hipMemcpy(maxhost, max, intSize, hipMemcpyDeviceToHost);

    hipFree(max);
    hipFree(check);

    return maxhost[0];
}

/********************* Find the Curl *****************************************/
int findCurl(int *sequence, int *table, int length, int capacity){
    int *tempResults;
    hipMalloc((void **) &tempResults, (length >> 1) * sizeof(int));
    int *cap;
    hipMalloc((void **) &cap, sizeof(int));
    hipMemcpy(cap, (int*)&capacity, sizeof(int), hipMemcpyHostToDevice);

    for(int i(0); i < (length >> 1); ++i) {
        //int *p = &(table[i][(length - 1) - i]);
        //findMin(p, length, &(tempResults[i]));
        findMin(table, i+1, (length - 1) - i, &(tempResults[i]), cap);
    }

    int *results = (int *) malloc((length >> 1) * sizeof(int));
    hipMemcpy(results, tempResults, (length >> 1) * sizeof(int), hipMemcpyDeviceToHost);
    for(int i(0); i < (length >> 1); ++i) {
        printf("%d ", results[i]);
    }
    printf("\n");

    int curl = findMax(tempResults, length);

    hipFree(tempResults);

    return curl;
}

void printTable(int *table, int length, int capacity) {
    int *CPUTable;
    CPUTable = (int *) malloc(capacity * capacity * sizeof(int));
    hipMemcpy(CPUTable, table, capacity * capacity * sizeof(int), hipMemcpyDeviceToHost);

    for(int i(0); i < length; ++i) {
        for(int j(0); j < length; ++j) {
            printf("%d ", CPUTable[(i * capacity) + j]);
        }
        printf("\n");
    }

    free(CPUTable);
}

__global__ void fillColumn(int *sequence, int *table, int *seqPosition, int *cap) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int index = *seqPosition;
    int capacity = *cap;
    int value = 1;
    
    if(row == index){}
    else if(sequence[index - (row + 1)] == sequence[index]) {
        int t = table[(row * capacity) + (index - (row + 1))];
        if(t == 0) {
            value = 2;
        } else {
            value = table[(row * capacity) + (index - (row + 1))] + 1;
        }
    }

    table[(row * capacity) + index] = value;
}

void initializeTable(int *sequence, int *table, int length, int capacity) {

    int *index;
    hipMalloc((void **)&index, sizeof(int));
    int *cap;
    hipMalloc((void **)&cap, sizeof(int));
    hipMemcpy(cap, (void *)&capacity, sizeof(int), hipMemcpyHostToDevice);

    for(int i(0); i < length; ++i) {
        hipMemcpy(index, (void *)&i, sizeof(int), hipMemcpyHostToDevice);
        fillColumn<<< dim3(i + 1, 1), 1 >>>(sequence, table, index, cap);
    }

    hipFree(index);
}

int main() {
    int *table;
    int capacity = INITIAL_CAPACITY;

    hipMalloc((void**)&table, (INITIAL_CAPACITY * INITIAL_CAPACITY) * sizeof(int));

    while (1) {

        hipMemset(table, 0, (capacity * capacity) * sizeof(int));
        
        char buffer[100];
        printf("Input a sequence to curl:\n");
        scanf("%s", buffer);

        int i(0);
        int sequence[INITIAL_CAPACITY];
        for (; buffer[i] != '\0'; ++i) {
            sequence[i] = buffer[i] - '0';
        }

        int arraySize = i;
        int *a;
        int iSize = arraySize * sizeof(int);
        hipMalloc((void**)&a, iSize);
        hipMemcpy(a, sequence, iSize, hipMemcpyHostToDevice);

        initializeTable(a, table, arraySize, capacity);
        printTable(table, arraySize, capacity);

        clock_t start = clock();

        int *size;
        hipMalloc((void**)&size, sizeof(int));
        int *cap;
        hipMalloc((void **)&cap, sizeof(int));
        hipMemcpy(cap, (void *)&capacity, sizeof(int), hipMemcpyHostToDevice);
        int curl = 0;

        while(curl != 1) {
            curl = findCurl(a, table, arraySize, capacity);
            sequence[arraySize] = curl;
            hipMemcpy(size, (int*)&arraySize, sizeof(int), hipMemcpyHostToDevice);
            iSize = ++arraySize * sizeof(int);
            hipMalloc((void**)&a, iSize);
            hipMemcpy(a, sequence, iSize, hipMemcpyHostToDevice);
            fillColumn<<< dim3(arraySize, 1), 1 >>>(a, table, size, cap);
        }

        clock_t stop = clock();
        double elapsed = ((double)(stop - start)) / CLOCKS_PER_SEC;
        printf("Elapsed time: %.3fs\n", elapsed);
        printf("curl is %d\n", curl);

        hipFree(a);
    }
    return 0;
}
