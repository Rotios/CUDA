#include "hip/hip_runtime.h"


#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#define INITIAL_CAPACITY 1024

/******************** Find the min value **************************/
__global__ void minCompare(int *a, int set, bool *check) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int tabx = idx * 1024 + set;
    int taby = idy * 1024 + set;

    if (idx == idy) { return; }

    int xval = a[tabx];
    int yval = a[taby];
    
    if (xval == 0 || xval == 1) {
        check[idx] = false;
    } else if (xval > yval) {
        check[idx] = false;
    }
}

__global__ void cudaMin(int *a, int set, bool *check, int* min) {
    int idx = blockIdx.x;

    if (check[idx]) {
        min[0] = a[idx * 1024 + set];
    }
}

/************************* Find the max value **********************/
__global__ void maxCompare(int *a, bool *check) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    if (idx == idy) { return; }

    int xval = a[idx];
    int yval = a[idy];

    if (xval < yval) {
        check[idx] = false;
    }
}

__global__ void cudaMax(int *a, bool *check, int* max) {
    int idx = blockIdx.x;

    if (check[idx]) {
        max[0] = a[idx];
    }
}

/*********************** Helper Methods ********************************************/
__global__ void cudaBoolFill(bool *arr, int length) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < length) {
        arr[i] = true;
    }
}

/********************** Min and Max Functions ******************************************/
void findMin(int *arr, const int length, const int offset, int *minimum) {
    bool *check;
    int *min;
    int *set;
    int one = 1;

    const int intSize = sizeof(int);
    const int bsize = length * sizeof(bool);

    hipMalloc((void**)&check, bsize);
    cudaBoolFill<<< dim3(length, 1), 1 >>>(check, length);

    hipMalloc((void**)&min, intSize);
    //cudaMemcpy(min, (void*)&one, intSize, cudaMemcpyHostToDevice);

    hipMalloc((void**)&set, intSize);

    hipMemcpy(set, (int*)&offset, intSize, hipMemcpyHostToDevice);

    minCompare<<< dim3(length, length), 1 >>>(arr, *set, check);
    cudaMin<<< dim3(length, 1), 1 >>>(arr, *set, check, min);

    int minhost[1];
    hipMemcpy(minhost, min, intSize, hipMemcpyDeviceToHost);

    hipFree(min);
    hipFree(check);

    hipMemcpy(minimum, (void *)&(minhost[0]), intSize, hipMemcpyHostToDevice);
    //minimum = minhost[0];
}

int findMax(int *arr, const int length) {
    bool *check;
    int *max;

    const int intSize = sizeof(int);
    const int bsize = length * sizeof(bool);

    hipMalloc((void**)&check, bsize);
    cudaBoolFill<<< dim3(length, 1), 1 >>>(check, length);

    hipMalloc((void**)&max, intSize);

    maxCompare<<< dim3(length, length), 1 >>>(arr, check);
    cudaMax<<< dim3(length, 1), 1 >>>(arr, check, max);

    int maxhost[1];
    hipMemcpy(maxhost, max, intSize, hipMemcpyDeviceToHost);

    hipFree(max);
    hipFree(check);

    return maxhost[0];
}

/********************* Find the Curl *****************************************/
int findCurl(int *sequence, int *table, int length){
    int *tempResults;
    hipMalloc((void **) &tempResults, (length >> 1) * sizeof(int));

    for(int i(0); i < (length >> 1); ++i) {
        //int *p = &(table[i][(length - 1) - i]);
        //findMin(p, length, &(tempResults[i]));
        findMin(table, i+1, (length - 1) - i, &(tempResults[i]));
    }
    int curl = findMax(tempResults, length);

    hipFree(tempResults);

    return curl;
}

void printTable(int *table, int length) {
    int *CPUTable;
    CPUTable = (int *) malloc(INITIAL_CAPACITY * INITIAL_CAPACITY * sizeof(int));
    hipMemcpy(CPUTable, table, INITIAL_CAPACITY * INITIAL_CAPACITY * sizeof(int), hipMemcpyDeviceToHost);

    for(int i(0); i < length; ++i) {
        for(int j(0); j < length; ++j) {
            printf("%d ", CPUTable[(i * 1024) + j]);
        }
        printf("\n");
    }

    free(CPUTable);
}

__global__ void fillColumn(int *sequence, int *table, int *seqPosition) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int index = *seqPosition;
    int value = 1;
    
    if(row == index){}
    else if(sequence[index - (row + 1)] == sequence[index]) {
        value = table[(row * 1024) + (index - (row + 1))] + 1;
    }

    table[(row * 1024) + index] = value;
}

void initializeTable(int *sequence, int *table, int length) {

    int *index;
    hipMalloc((void **)&index, sizeof(int *));

    for(int i(0); i < length; ++i) {
        hipMemcpy(index, (void *)&i, sizeof(int), hipMemcpyHostToDevice);
        fillColumn <<< dim3(i + 1, 1), 1 >>>(sequence, table, index);
        //printTable(table, i);
    }

    hipFree(index);
}

int main()
{
    int *table;

    hipMalloc((void**)&table, (INITIAL_CAPACITY * INITIAL_CAPACITY) * sizeof(int));

    while (1) {

        hipMemset(table, 0, (INITIAL_CAPACITY * INITIAL_CAPACITY) * sizeof(int));
        
        char buffer[100];

        printf("Input a sequence to curl:\n");
        scanf("%s", buffer);

        int i(0);
        int sequence[100];

        for (; buffer[i] != '\0'; ++i) {
            sequence[i] = buffer[i] - '0';
        }

        int arraySize = i;

        int *a;
        int iSize = arraySize * sizeof(int);
        hipMalloc((void**)&a, iSize);
        hipMemcpy(a, sequence, iSize, hipMemcpyHostToDevice);

        printTable(table, arraySize);

        initializeTable(a, table, arraySize);

        printTable(table, arraySize);

        clock_t start = clock();

        int curl = findCurl(a, table, arraySize);

        clock_t stop = clock();
        double elapsed = ((double)(stop - start)) / CLOCKS_PER_SEC;
        printf("Elapsed time: %.3fs\n", elapsed);

        printf("curl is %d\n", curl);

        hipFree(a);
    }
    return 0;
}
